#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <cxxopts.hpp>
#include <iostream>
#include <stdio.h>

typedef int64_t i64;
typedef i64 field_elem[16];
typedef unsigned char u8;

__device__ void toBase64(const u8 in[32], char out[45]) {
    static const char* b64chars =
        "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789+/";

    size_t i, j, v;

    for (i = 0, j = 0; i < 32; i += 3, j += 4) {
        v = in[i];
        v = i + 1 < 32 ? v << 8 | in[i + 1] : v << 8;
        v = i + 2 < 32 ? v << 8 | in[i + 2] : v << 8;

        out[j] = b64chars[(v >> 18) & 0x3F];
        out[j + 1] = b64chars[(v >> 12) & 0x3F];
        if (i + 1 < 32) {
            out[j + 2] = b64chars[(v >> 6) & 0x3F];
        } else {
            out[j + 2] = '=';
        }
        if (i + 2 < 32) {
            out[j + 3] = b64chars[v & 0x3F];
        } else {
            out[j + 3] = '=';
        }
    }
    out[44] = '\0';
}

// https://martin.kleppmann.com/papers/curve25519.pdf

__device__ void unpack25519(field_elem out, const u8 in[32]) {
    int i;
    for (i = 0; i < 16; i++) {
        out[i] = in[2 * i] + ((int64_t)in[2 * i + 1] << 8);
    }
    out[15] &= 0x7fff;
}

__device__ void swap25519(field_elem p, field_elem q, int bit) {
    i64 t, i, c = ~(bit - 1);
    for (i = 0; i < 16; i++) {
        t = c & (p[i] ^ q[i]);
        p[i] ^= t;
        q[i] ^= t;
    }
}

__device__ void field_elem_add(field_elem out, const field_elem a, const field_elem b) {
    for (int i = 0; i < 16; i++) {
        out[i] = a[i] + b[i];
    }
}

__device__ void field_elem_sub(field_elem out, const field_elem a, const field_elem b) {
    for (int i = 0; i < 16; i++) {
        out[i] = a[i] - b[i];
    }
}

__device__ void carry25519(field_elem elem) {
    int i;
    i64 carry;
    for (i = 0; i < 16; ++i) {
        carry = elem[i] >> 16;
        elem[i] -= carry << 16;
        if (i < 15) {
            elem[i + 1] += carry;
        } else {
            elem[0] += 38 * carry;
        }
    }
}

__device__ void field_elem_mul(field_elem out, const field_elem a, const field_elem b) {
    i64 i, j, product[31];
    for (i = 0; i < 31; i++) {
        product[i] = 0;
    }
    for (i = 0; i < 16; i++) {
        for (j = 0; j < 16; j++) {
            product[i + j] += a[i] * b[j];
        }
    }
    for (i = 0; i < 15; i++) {
        product[i] += 38 * product[i + 16];
    }
    for (i = 0; i < 16; i++) {
        out[i] = product[i];
    }
    carry25519(out);
    carry25519(out);
}

__device__ void field_elem_inv(field_elem out, const field_elem in) {
    field_elem c;
    int i;
    for (i = 0; i < 16; ++i) {
        c[i] = in[i];
    }
    for (i = 253; i >= 0; --i) {
        field_elem_mul(c, c, c);
        if (i != 2 && i != 4) {
            field_elem_mul(c, c, in);
        }
    }
    for (i = 0; i < 16; ++i) {
        out[i] = c[i];
    }
}

__device__ void pack25519(u8 out[32], const field_elem in) {
    int i, j, carry;
    field_elem m, t;
    for (i = 0; i < 16; ++i) {
        t[i] = in[i];
    }
    carry25519(t);
    carry25519(t);
    carry25519(t);
    for (j = 0; j < 2; ++j) {
        m[0] = t[0] - 0xffed;
        for (i = 1; i < 15; ++i) {
            m[i] = t[i] - 0xffff - ((m[i - 1] >> 16) & 1);
            m[i - 1] &= 0xffff;
        }
        m[15] = t[15] - 0x7fff - ((m[14] >> 16) & 1);
        carry = (m[15] >> 16) & 1;
        m[14] &= 0xffff;
        swap25519(t, m, 1 - carry);
    }
    for (i = 0; i < 16; ++i) {
        out[2 * i] = t[i] & 0xff;
        out[2 * i + 1] = t[i] >> 8;
    }
}

__device__ void genpub(const u8 privkey[32], u8 pub[32]) {
    static const field_elem _121665 = { 0xDB41, 1 };
    static const u8 _9[32] = { 9 };

    u8 clamped[32];
    i64 bit, i;
    field_elem a, b, c, d, e, f, x;
    for (i = 0; i < 32; i++) {
        clamped[i] = privkey[i];
    }
    clamped[0] &= 0xf8;
    clamped[31] = (clamped[31] & 0x7f) | 0x40;
    unpack25519(x, _9);
    for (i = 0; i < 16; ++i) {
        b[i] = x[i];
        d[i] = a[i] = c[i] = 0;
    }
    a[0] = d[0] = 1;
    for (i = 254; i >= 0; --i) {
        bit = (clamped[i >> 3] >> (i & 7)) & 1;
        swap25519(a, b, bit);
        swap25519(c, d, bit);
        field_elem_add(e, a, c);
        field_elem_sub(a, a, c);
        field_elem_add(c, b, d);
        field_elem_sub(b, b, d);
        field_elem_mul(d, e, e);
        field_elem_mul(f, a, a);
        field_elem_mul(a, c, a);
        field_elem_mul(c, b, e);
        field_elem_add(e, a, c);
        field_elem_sub(a, a, c);
        field_elem_mul(b, a, a);
        field_elem_sub(c, d, f);
        field_elem_mul(a, c, _121665);
        field_elem_add(a, a, d);
        field_elem_mul(c, c, a);
        field_elem_mul(a, d, f);
        field_elem_mul(d, b, x);
        field_elem_mul(b, e, e);
        swap25519(a, b, bit);
        swap25519(c, d, bit);
    }
    field_elem_inv(c, c);
    field_elem_mul(a, a, c);
    pack25519(pub, a);
}

__device__ char cutolower(char c) {
    if (c >= 'A' && c <= 'Z') {
        return c + 32;
    }
    return c;
}

__device__ size_t custrlen(const char* s) {
    size_t i = 0;
    while (s[i] != '\0') {
        i++;
    }
    return i;
}

__device__ int custrncasecmp(const char* s1, const char* s2, size_t n) {
    for (size_t i = 0; i < n; i++) {
        if (cutolower(s1[i]) != cutolower(s2[i])) {
            return 1;
        }
    }
    return 0;
}

__device__ char* custrcasestr(const char* haystack, const char* needle) {
    char c, sc;
    size_t len;
    if ((c = *needle++) != 0) {
        c = cutolower((unsigned char)c);
        len = custrlen(needle);
        do {
            do {
                if ((sc = *haystack++) == 0) {
                    return (NULL);
                }
            } while ((char)cutolower((unsigned char)sc) != c);
        } while (custrncasecmp(haystack, needle, len) != 0);
        haystack--;
    }
    return ((char*)haystack);
}

__global__ void generator(const char* _needle, size_t needlesize, int in) {
    int tid = threadIdx.x, idx = blockIdx.x * blockDim.x + tid;

    extern __shared__ char needle[];

    if (tid < needlesize) {
        needle[tid] = _needle[tid];
    } else if (tid == needlesize) {
        needle[tid] = '\0';
    }

    __syncthreads();

    hiprandState_t state;
    hiprand_init(0, idx, 0, &state);

    u8 privkey[32], pubkey[32];
    char privkey_b64[45], pubkey_b64[45];

    while (true) {
        for (int i = 0; i < 32; i++) {
            privkey[i] = hiprand(&state) % 256;
        }

        genpub(privkey, pubkey);

        toBase64(pubkey, pubkey_b64);

        char save = pubkey_b64[in];
        pubkey_b64[in] = '\0';
        if (custrcasestr(pubkey_b64, needle) != NULL) {
            pubkey_b64[in] = save;
            toBase64(privkey, privkey_b64);
            printf("private: %s | public: %s\n", privkey_b64, pubkey_b64);
        }
    }
}

int main(int argc, char** argv) {
    cxxopts::Options options(
        "wicuvanity",
        "Generate wireguard private and public keys with a specific public key prefix"
    );

    options.add_options()("needle", "needle to find", cxxopts::value<std::string>())
                       ("in", "needle in first ... characters", cxxopts::value<int>()->default_value("10"))
                       ("gridsize", "Number of blocks", cxxopts::value<int>()->default_value("1024"))
                       ("blocksize", "Number of threads in block", cxxopts::value<int>()->default_value("256"))
                       ("h,help", "print usage");

    options.parse_positional({ "needle" });
    options.positional_help("needle");

    auto result = options.parse(argc, argv);

    if (result.count("help")) {
        std::cout << options.help() << std::endl;
        return 0;
    }

    auto needle = result["needle"].as<std::string>();
    char *pndl_host = const_cast<char*>(needle.c_str()), *pndl_dev;

    hipHostRegister(pndl_host, needle.size(), hipHostRegisterReadOnly);
    hipHostGetDevicePointer(&pndl_dev, pndl_host, 0);

    int gridsize = result["gridsize"].as<int>(), blocksize = result["blocksize"].as<int>(),
        needlesize = needle.size(), in = result["in"].as<int>();

    if (blocksize < needlesize) {
        std::cerr << "Error: block size must be greater than needle size" << std::endl;
        return 1;
    }

    generator<<<gridsize, blocksize, needlesize + 1>>>(pndl_dev, needlesize, in);
    hipDeviceSynchronize();

    hipHostUnregister(pndl_host);

    hipError_t code;
    switch (code = hipGetLastError()) {
        case hipSuccess:
            return 0;
        case hipErrorLaunchOutOfResources:
            std::cerr << "Error: out of resources. Try decreasing --blocksize to " << blocksize - 32
                      << std::endl;
            return 1;
        default:
            std::cerr << "Error: " << hipGetErrorString(code) << std::endl;
            return 1;
    }
}
